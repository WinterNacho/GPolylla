#include "hip/hip_runtime.h"
/* Polygon mesh generator
//POSIBLE BUG: el algoritmo no viaja por todos los halfedges dentro de un poligono, 
    //por lo que pueden haber semillas que no se borren y tener poligonos repetidos de output
*/

#ifndef POLYLLA_HPP
#define POLYLLA_HPP


#include <array>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <cmath>
#include "kernel.cu"
#include <chrono>
#include <iomanip>


#define print_e(eddddge) eddddge<<" ( "<<mesh_input->origin(eddddge)<<" - "<<mesh_input->target(eddddge)<<") "

class Polylla
{
private:
    typedef std::vector<int> _polygon; 
    typedef std::vector<char> bit_vector; 


    Triangulation *mesh_input; // Halfedge triangulation
    Triangulation *mesh_output;
    std::vector<int> output_seeds; //Seeds of the polygon

    //std::vector<int> triangles; //True if the edge generated a triangle CHANGE!!!!

    bit_vector max_edges; //True if the edge i is a max edge
    bit_vector frontier_edges; //True if the edge i is a frontier edge
    std::vector<int> seed_edges; //Seed edges that generate polygon simple and non-simple

    // Auxiliary array used during the barrier-edge elimination
    std::vector<int> triangle_list;
    bit_vector seed_bet_mark;

    //Statistics
    int m_polygons = 0; //Number of polygons
    int n_frontier_edges = 0; //Number of frontier edges
    int n_barrier_edge_tips = 0; //Number of barrier edge tips
    int n_polygons_to_repair = 0;
    int n_polygons_added_after_repair = 0;

    // Times
    double t_label_max_edges = 0;
    double t_label_frontier_edges = 0;
    double t_label_seed_edges = 0;
    double t_traversal_and_repair = 0;
    double t_traversal = 0;
    double t_repair = 0;
    
public:

    Polylla() {}; //Default constructor

    //Constructor with triangulation
    Polylla(Triangulation *input_mesh){
        this->mesh_input = input_mesh;
        construct_Polylla();
    }



    //Constructor from a OFF file
    Polylla(std::string off_file){

        this->mesh_input = new Triangulation(off_file);
        mesh_output = new Triangulation(*mesh_input);
        construct_Polylla();
    }

    //Constructor from a node_file, ele_file and neigh_file
    Polylla(std::string node_file, std::string ele_file, std::string neigh_file){
        this->mesh_input = new Triangulation(node_file, ele_file, neigh_file);
        //call copy constructor
        mesh_output = new Triangulation(*mesh_input);
        construct_Polylla();
    }

    ~Polylla() {
        //triangles.clear(); 
        max_edges.clear(); 
        frontier_edges.clear();
        seed_edges.clear(); 
        seed_bet_mark.clear();
        triangle_list.clear();
        delete mesh_input;
        delete mesh_output;
    }

    void construct_Polylla(){

        max_edges = bit_vector(mesh_input->halfEdges(), false);
        frontier_edges = bit_vector(mesh_input->halfEdges(), false);
        //triangles = mesh_input->get_Triangles(); //Change by triangle list
        seed_bet_mark = bit_vector(this->mesh_input->halfEdges(), false);

        //terminal_edges = bit_vector(mesh_input->halfEdges(), false);
        //seed_edges = bit_vector(mesh_input->halfEdges(), false);
        
        // copy to device and initialize
        auto t_start = std::chrono::high_resolution_clock::now();

        // declare and initialize device arrays
        //int *d_triangles;
        int n_triangle = mesh_input->faces();

        // copy halfedges to device
        int n_halfedges = mesh_input->halfEdges();
        halfEdge *halfedges_d, *halfedges_h = new halfEdge[n_halfedges];
        halfedges_h = mesh_input->HalfEdges.data();
        hipMalloc(&halfedges_d, n_halfedges*sizeof(halfEdge) );
        hipMemcpy(halfedges_d, halfedges_h, n_halfedges*sizeof(halfEdge), hipMemcpyHostToDevice );

        // copy vertices to device
        int n_vertices = mesh_input->vertices();
        vertex *vertices_d, *vertices_h = new vertex[n_vertices];
        vertices_h = mesh_input->Vertices.data();
        hipMalloc(&vertices_d, n_vertices*sizeof(vertex) );
        hipMemcpy(vertices_d, vertices_h, n_vertices*sizeof(vertex), hipMemcpyHostToDevice );

        bit_vector_d *max_edges_d;
        hipMalloc(&max_edges_d, n_halfedges*sizeof(bit_vector_d) );
        hipMemset(max_edges_d, 0, n_halfedges*sizeof(bit_vector_d));

        // DEFINE GRID AND BLOCK SIZE
        dim3 block, grid;
        block = dim3(BSIZE, 1, 1);    
        grid = dim3((n_triangle + BSIZE - 1)/BSIZE, 1, 1);


        auto t_end = std::chrono::high_resolution_clock::now();
        double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"Copy vectors to device in "<<elapsed_time_ms<<" ms"<<std::endl;



        gpuErrchk( hipDeviceSynchronize() );
        //Label max edges of each triangle
        //for (size_t t = 0; t < mesh_input->faces(); t++){
        t_start = std::chrono::high_resolution_clock::now();

        // GPU
        label_edges_max_d<<<(n_triangle + BSIZE - 1)/BSIZE, block>>>(max_edges_d, vertices_d, halfedges_d, n_triangle);
        gpuErrchk( hipDeviceSynchronize() );

        // CPU
        /*for(int i = 0; i < mesh_input->faces(); i++)
            max_edges[label_max_edge(mesh_input->incident_halfedge(i))] = true;//*/

        
        // copy back to host
        /*bit_vector_d *h_max_edges = new bit_vector_d[n_halfedges];
        hipMemcpy( h_max_edges, max_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        //hipMemcpy( max_edges, max_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        gpuErrchk( hipDeviceSynchronize() );

        int count = 0;
        for(int i = 0; i < n_halfedges; i++){
            //printf("%i %i %i\n", i, (int) h_max_edges[i], (int) max_edges[i]);
            //assert(max_edges[i] == h_max_edges[i]);
            if (max_edges[i] != h_max_edges[i]){
                //printf("%i %i %i\n", i, (int) h_max_edges[i], (int) max_edges[i]);
                //printf ("Distances: %f %f %f\n", mesh_input->distance(i), mesh_input->distance(mesh_input->next(i)), mesh_input->distance(mesh_input->prev(i)));
                count++;
            }
        }
        printf ("Number of errors: %i\n", count); //*/

        
        //for(int i = 0; i < n_halfedges; i++)
        //    max_edges[i] = h_max_edges[i];

        
        t_end = std::chrono::high_resolution_clock::now();
        elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"Labered max edges in "<<elapsed_time_ms<<" ms"<<std::endl;



        t_start = std::chrono::high_resolution_clock::now();

        // GPU
        //block = dim3(BSIZE, 1, 1);    
        //grid = dim3((n_halfedges + BSIZE - 1)/BSIZE, 1, 1);    

        bit_vector_d *frontier_edges_d;
        hipMalloc(&frontier_edges_d, sizeof(bit_vector_d)*n_halfedges);
        label_phase<<<(n_halfedges + BSIZE - 1)/BSIZE,block>>>(halfedges_d, max_edges_d, frontier_edges_d, n_halfedges); 
        hipDeviceSynchronize(); //*/


        /* // CPU
        //Label frontier edges
        for (std::size_t e = 0; e < mesh_input->halfEdges(); e++){
            if(is_frontier_edge(e)){
                frontier_edges[e] = true;
                n_frontier_edges++;
            }
        }
        gpuErrchk( hipDeviceSynchronize() );
        // copy back to host
        bit_vector_d *h_frontier_edges = new bit_vector_d[n_halfedges];
        hipMemcpy( h_frontier_edges, frontier_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        //hipMemcpy( h_frontier_edges, frontier_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        gpuErrchk( hipDeviceSynchronize() );
        

        for(int i = 0; i < n_halfedges; i++){
            //printf("%i %i %i\n", i, (int) h_max_edges[i], (int) max_edges[i]);
            assert(max_edges[i] == h_max_edges[i]);
            //printf("%i %i %i\n", i, (int) h_frontier_edges[i], (int) frontier_edges[i]);
        }
        
        for(int i = 0; i < n_halfedges; i++)
            frontier_edges[i] = h_frontier_edges[i];//*/

        t_end = std::chrono::high_resolution_clock::now();
        elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"Labeled frontier edges in "<<elapsed_time_ms<<" ms"<<std::endl;
        

        t_start = std::chrono::high_resolution_clock::now();

        // GPU SEED PHASE
        half *seed_edges_ad;
        int *seed_edges_d;
        hipMalloc(&seed_edges_ad, sizeof(half)*n_halfedges);
        hipMemset(seed_edges_ad, 0, sizeof(half)*n_halfedges);
        hipMalloc(&seed_edges_d, sizeof(int)*n_halfedges);
        //gpuErrchk( hipDeviceSynchronize() );
        seed_phase_d<<<(n_halfedges + BSIZE - 1)/BSIZE,block>>>(halfedges_d, max_edges_d, seed_edges_ad, n_halfedges); 
        gpuErrchk( hipDeviceSynchronize() );

        int seed_len;
        scan_parallel_tc_2<int>(seed_edges_d, seed_edges_ad, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );
        hipMemcpy( &seed_len, seed_edges_d + n_halfedges - 1, sizeof(int), hipMemcpyDeviceToHost );
        //int seed_len = scan(seed_edges_d, seed_edges_ad, n_halfedges); // ESTO SE PUEDE MEJORAR!
        //gpuErrchk( hipDeviceSynchronize() );
        //printf ("-> %i %i %i %i\n", grid.x, grid.y, grid.z, (n_halfedges + BSIZE - 1)/BSIZE);
        compaction_d<<<(n_halfedges + BSIZE - 1)/BSIZE,BSIZE>>>(seed_edges_d, seed_edges_d, seed_edges_ad, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );
        //compaction_cub(seed_edges_d, d_num, max_edges_d, seed_edges_ad, n_halfedges);
        //gpuErrchk( hipDeviceSynchronize() );
        //printf("\ndone GPU seed phase....\n\n");//*/


        /*//label seeds edges,
        for (std::size_t e = 0; e < mesh_input->halfEdges(); e++)
            if(mesh_input->is_interior_face(e) && is_seed_edge(e))
                seed_edges.push_back(e);
        t_end = std::chrono::high_resolution_clock::now();

        // copy back to host
        int *h_seed_edges = new int[n_halfedges];
        gpuErrchk( hipDeviceSynchronize() );
        hipMemcpy( h_seed_edges, seed_edges_d, seed_len*sizeof(int), hipMemcpyDeviceToHost );
        gpuErrchk( hipDeviceSynchronize() );

        printf("Number of seeds: %i\n", seed_len);
        for (int i = 0; i < seed_len; i++){
            if (seed_edges[i] != h_seed_edges[i])
                printf ("%i %i %i\n", i, h_seed_edges[i], seed_edges[i]);
            //assert(h_seed_edges[i] == seed_edges[i]);
        }//

        for (int i = 0; i < seed_len; i++)
            seed_edges[i] = h_seed_edges[i]; */


        t_end = std::chrono::high_resolution_clock::now();
        elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"Labeled seed edges in "<<elapsed_time_ms<<" ms"<<std::endl;


        t_start = std::chrono::high_resolution_clock::now();
        // GPU
        // TRAVEL PHASE IN GPU!!!!
        int *output_seed_d;
        halfEdge *output_HalfEdges_d;
        hipMalloc(&output_HalfEdges_d, sizeof(halfEdge)*n_halfedges);
        travel_phase_d<<<(n_halfedges + BSIZE - 1)/BSIZE,block>>>(output_HalfEdges_d, halfedges_d, max_edges_d, frontier_edges_d, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );
        hipMalloc(&output_seed_d , sizeof(int)*seed_len);
        search_frontier_edge_d<<<(seed_len+BSIZE-1)/BSIZE,block>>>(output_seed_d, halfedges_d, frontier_edges_d, seed_edges_d, seed_len);
        gpuErrchk( hipDeviceSynchronize() );

 

        /*//Travel phase: Generate polygon mesh
        int polygon_seed;
        //Foreach seed edge generate polygon
        for(auto &e : seed_edges){
            polygon_seed = travel_triangles(e);
            if(!has_BarrierEdgeTip(polygon_seed)){ //If the polygon is a simple polygon then is part of the mesh
                output_seeds.push_back(polygon_seed);
            }else{ //Else, the polygon is send to reparation phase
                auto t_start_repair = std::chrono::high_resolution_clock::now();
                barrieredge_tip_reparation(polygon_seed);
                auto t_end_repair = std::chrono::high_resolution_clock::now();
                t_repair += std::chrono::duration<double, std::milli>(t_end_repair-t_start_repair).count();
            }         
        }    

        //printf ("aca\n");
        // COPY OUTPUT_SEED TO HOST
        int *output_seed_h = new int[seed_len];
        hipMemcpy(output_seed_h, output_seed_d, sizeof(int)*seed_len, hipMemcpyDeviceToHost);
        for (uint i = 0; i < seed_len; i++) {
            printf("%i %i %i\n",i,output_seed_h[i],output_seeds[i]);
            output_seeds[i] == output_seed_h[i];
            //assert(output_seed_h[i] == output_seeds[i]);
        }*/

        t_end = std::chrono::high_resolution_clock::now();
        t_traversal = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"Traversal phase in "<<t_traversal<<" ms"<<std::endl;

        t_start = std::chrono::high_resolution_clock::now();
        // back to host
        bit_vector_d *h_max_edges = new bit_vector_d[n_halfedges];
        hipMemcpy( h_max_edges, max_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        gpuErrchk( hipDeviceSynchronize() );
        bit_vector_d *h_frontier_edges = new bit_vector_d[n_halfedges];
        hipMemcpy( h_frontier_edges, frontier_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        int *h_seed_edges = new int[n_halfedges];
        gpuErrchk( hipDeviceSynchronize() );
        hipMemcpy( h_seed_edges, seed_edges_d, seed_len*sizeof(int), hipMemcpyDeviceToHost );
        int *output_seed_h = new int[seed_len];
        hipMemcpy(output_seed_h, output_seed_d, sizeof(int)*seed_len, hipMemcpyDeviceToHost);
        gpuErrchk( hipDeviceSynchronize() );


        t_end = std::chrono::high_resolution_clock::now();
        elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"Back to host in "<<elapsed_time_ms<<" ms"<<std::endl;

        // standard output, time not measured
        for (int i = 0; i < n_halfedges; i++)
            max_edges[i] = h_max_edges[i];
        for (int i = 0; i < n_halfedges; i++)
            frontier_edges[i] = h_frontier_edges[i];
        //for (int i = 0; i < seed_len; i++)
        //    seed_edges[i] = h_seed_edges[i];
        //for (int i = 0; i < seed_len; i++)
        //    output_seeds[i] = output_seed_h[i];
        std::vector<int> aux_seed_edges(h_seed_edges, h_seed_edges + seed_len);
        std::vector<int> aux_output(output_seed_h, output_seed_h + seed_len);
        seed_edges = aux_seed_edges;
        output_seeds = aux_output;
        gpuErrchk( hipDeviceSynchronize() );

        this->m_polygons = output_seeds.size();

        // repair phase
        int polygon_seed;
        //Foreach seed edge generate polygon
        for(auto &e : seed_edges){
            polygon_seed = travel_triangles(e);

            if(has_BarrierEdgeTip(polygon_seed)){ //the polygon is send to reparation phase
                printf("aca\n");
                auto t_start_repair = std::chrono::high_resolution_clock::now();
                barrieredge_tip_reparation(polygon_seed);
                output_seeds.push_back(polygon_seed);
                auto t_end_repair = std::chrono::high_resolution_clock::now();
                t_repair += std::chrono::duration<double, std::milli>(t_end_repair-t_start_repair).count();
            }//*/         
        }
        std::cout<<"CPU repair phase in "<<t_repair<<" ms"<<std::endl;

        // time
        t_traversal_and_repair = t_traversal + t_repair;

        int count = 0;
        for (std::size_t e = 0; e < mesh_input->halfEdges(); e++){
            if(is_frontier_edge(e)){
                printf("%i\n",frontier_edges[count]);
                n_frontier_edges++;
                count++;
            }
        }

        std::cout<<"Mesh with "<<m_polygons<<" polygons "<<n_frontier_edges/2<<" edges and "<<n_barrier_edge_tips<<" barrier-edge tips."<<std::endl;
        //mesh_input->print_pg(std::to_string(mesh_input->vertices()) + ".pg");    
        
    }


    void print_stats(std::string filename){
        //Time
        std::cout<<"Time to generate Triangulation: "<<mesh_input->get_triangulation_generation_time()<<" ms"<<std::endl;
        std::cout<<"Time to label max edges "<<t_label_max_edges<<" ms"<<std::endl;
        std::cout<<"Time to label frontier edges "<<t_label_frontier_edges<<" ms"<<std::endl;
        std::cout<<"Time to label seed edges "<<t_label_seed_edges<<" ms"<<std::endl;
        std::cout<<"Time to label total"<<t_label_max_edges+t_label_frontier_edges+t_label_seed_edges<<" ms"<<std::endl;
        std::cout<<"Time to traversal and repair "<<t_traversal_and_repair<<" ms"<<std::endl;
        std::cout<<"Time to traversal "<<t_traversal<<" ms"<<std::endl;
        std::cout<<"Time to repair "<<t_repair<<" ms"<<std::endl;
        std::cout<<"Time to generate polygonal mesh "<<t_label_max_edges + t_label_frontier_edges + t_label_seed_edges + t_traversal_and_repair<<" ms"<<std::endl;

        //Memory
        long long m_max_edges =  sizeof(decltype(max_edges.back())) * max_edges.capacity();
        long long m_frontier_edge = sizeof(decltype(frontier_edges.back())) * frontier_edges.capacity();
        long long m_seed_edges = sizeof(decltype(seed_edges.back())) * seed_edges.capacity();
        long long m_seed_bet_mar = sizeof(decltype(seed_bet_mark.back())) * seed_bet_mark.capacity();
        long long m_triangle_list = sizeof(decltype(triangle_list.back())) * triangle_list.capacity();
        long long m_mesh_input = mesh_input->get_size_vertex_half_edge();
        long long m_mesh_output = mesh_output->get_size_vertex_half_edge();
        long long m_vertices_input = mesh_input->get_size_vertex_struct();
        long long m_vertices_output = mesh_output->get_size_vertex_struct();

        std::ofstream out(filename);
        std::cout<<"Printing JSON file as "<<filename<<std::endl;
        out<<"{"<<std::endl;
        out<<"\"n_polygons\": "<<m_polygons<<","<<std::endl;
        out<<"\"n_frontier_edges\": "<<n_frontier_edges/2<<","<<std::endl;
        out<<"\"n_barrier_edge_tips\": "<<n_barrier_edge_tips<<","<<std::endl;
        out<<"\"n_half_edges\": "<<mesh_input->halfEdges()<<","<<std::endl;
        out<<"\"n_faces\": "<<mesh_input->faces()<<","<<std::endl;
        out<<"\"n_vertices\": "<<mesh_input->vertices()<<std::endl;
        out<<"\"n_polygons_to_repair\": "<<n_polygons_to_repair<<","<<std::endl;
        out<<"\"n_polygons_added_after_repair\": "<<n_polygons_added_after_repair<<","<<std::endl;
        out<<"\"time_triangulation_generation\": "<<mesh_input->get_triangulation_generation_time()<<","<<std::endl;
        out<<"\"time_to_label_max_edges\": "<<t_label_max_edges<<","<<std::endl;
        out<<"\"time_to_label_frontier_edges\": "<<t_label_frontier_edges<<","<<std::endl;
        out<<"\"time_to_label_seed_edges\": "<<t_label_seed_edges<<","<<std::endl;
        out<<"\"time_to_label_total\": "<<t_label_max_edges+t_label_frontier_edges+t_label_seed_edges<<","<<std::endl;
        out<<"\"time_to_traversal_and_repair\": "<<t_traversal_and_repair<<","<<std::endl;
        out<<"\"time_to_traversal\": "<<t_traversal<<","<<std::endl;
        out<<"\"time_to_repair\": "<<t_repair<<","<<std::endl;
        out<<"\"time_to_generate_polygonal_mesh\": "<<t_label_max_edges + t_label_frontier_edges + t_label_seed_edges + t_traversal_and_repair<<","<<std::endl;
        out<<"\t\"memory_max_edges\": "<<m_max_edges<<","<<std::endl;
        out<<"\t\"memory_frontier_edge\": "<<m_frontier_edge<<","<<std::endl;
        out<<"\t\"memory_seed_edges\": "<<m_seed_edges<<","<<std::endl;
        out<<"\t\"memory_seed_bet_mar\": "<<m_seed_bet_mar<<","<<std::endl;
        out<<"\t\"memory_triangle_list\": "<<m_triangle_list<<","<<std::endl;
        out<<"\t\"memory_mesh_input\": "<<m_mesh_input<<","<<std::endl;
        out<<"\t\"memory_mesh_output\": "<<m_mesh_output<<","<<std::endl;
        out<<"\t\"memory_vertices_input\": "<<m_vertices_input<<","<<std::endl;
        out<<"\t\"memory_vertices_output\": "<<m_vertices_output<<","<<std::endl;
        out<<"\t\"memory_total\": "<<m_max_edges + m_frontier_edge + m_seed_edges + m_seed_bet_mar + m_triangle_list + m_mesh_input + m_mesh_output + m_vertices_input + m_vertices_output<<std::endl;
        out<<"}"<<std::endl;
        out.close();
    }




    //function whose input is a vector and print the elements of the vector
    void print_vector(std::vector<int> &vec){
        std::cout<<vec.size()<<" ";
        for (auto &v : vec){
            std::cout << v << " ";
        }
        std::cout << std::endl;
    }

    //Print ale file of the polylla mesh
    void print_ALE(std::string filename){
        std::ofstream out(filename);
        _polygon poly;
        out<<"# domain type\nCustom\n";
        out<<"# nodal coordinates: number of nodes followed by the coordinates \n";
        out<<mesh_input->vertices()<<std::endl;
        //print nodes
        for(std::size_t v = 0; v < mesh_input->vertices(); v++)
            out<<std::setprecision(15)<<mesh_input->get_PointX(v)<<" "<<mesh_input->get_PointY(v)<<std::endl; 
        out<<"# element connectivity: number of elements followed by the elements\n";
        out<<this->m_polygons<<std::endl;
        //print polygons
        int size_poly;
        int e_curr;
        for(auto &e_init : output_seeds){
            size_poly = 1;
            e_curr = mesh_output->next(e_init);
            while(e_init != e_curr){
                size_poly++;
                e_curr = mesh_output->next(e_curr);
            }
            out<<size_poly<<" ";            

            out<<mesh_output->origin(e_init)<<" ";
            e_curr = mesh_output->next(e_init);
            while(e_init != e_curr){
                out<<mesh_output->origin(e_curr)<<" ";
                e_curr = mesh_output->next(e_curr);
            }
            out<<std::endl; 
        }
        //Print borderedges
        out<<"# indices of nodes located on the Dirichlet boundary\n";
        ///Find borderedges
        int b_curr, b_init = 0;
        for(std::size_t i = mesh_input->halfEdges()-1; i != 0; i--){
            if(mesh_input->is_border_face(i)){
                b_init = i;
                break;
            }
        }
        out<<mesh_input->origin(b_init)<<" ";
        b_curr = mesh_input->prev(b_init);
        while(b_init != b_curr){
            out<<mesh_input->origin(b_curr)<<" ";
            b_curr = mesh_input->prev(b_curr);
        }
        out<<std::endl;
        out<<"# indices of nodes located on the Neumann boundary\n0\n";
        out<<"# xmin, xmax, ymin, ymax of the bounding box\n";
        double xmax = mesh_input->get_PointX(0);
        double xmin = mesh_input->get_PointX(0);
        double ymax = mesh_input->get_PointY(0);
        double ymin = mesh_input->get_PointY(0);
        //Search min and max coordinates
        for(std::size_t v = 0; v < mesh_input->vertices(); v++){
            //search range x
            if(mesh_input->get_PointX(v) > xmax )
                xmax = mesh_input->get_PointX(v);
            if(mesh_input->get_PointX(v) < xmin )
                xmin = mesh_input->get_PointX(v);
            //search range y
            if(mesh_input->get_PointY(v) > ymax )
                ymax = mesh_input->get_PointY(v);
            if(mesh_input->get_PointY(v) < ymin )
                ymin = mesh_input->get_PointY(v);
        }
        out<<xmin<<" "<<xmax<<" "<<ymin<<" "<<ymax<<std::endl;
        out.close();
    }

    //Print off file of the polylla mesh
    void print_OFF(std::string filename){
        std::ofstream out(filename);

      //  out<<"{ appearance  {+edge +face linewidth 2} LIST\n";
        out<<"OFF"<<std::endl;
        //num_vertices num_polygons 0
        out<<std::setprecision(15)<<mesh_input->vertices()<<" "<<m_polygons<<" 0"<<std::endl;
        //print nodes
        for(std::size_t v = 0; v < mesh_input->vertices(); v++)
            out<<mesh_input->get_PointX(v)<<" "<<mesh_input->get_PointY(v)<<" 0"<<std::endl; 
        //print polygons
        int size_poly;
        int e_curr;
        for(auto &e_init : output_seeds){
            size_poly = 1;
            e_curr = mesh_output->next(e_init);
            while(e_init != e_curr){
                size_poly++;
                e_curr = mesh_output->next(e_curr);
            }
            out<<size_poly<<" ";            

            out<<mesh_output->origin(e_init)<<" ";
            e_curr = mesh_output->next(e_init);
            while(e_init != e_curr){
                out<<mesh_output->origin(e_curr)<<" ";
                e_curr = mesh_output->next(e_curr);
            }
            out<<std::endl; 
        }
      //  out<<"}"<<std::endl;
        out.close();
    }

    //Print a halfedge file
    //The first line of the file is the number of halfedges
    //The rest of the lines are the halfedges with the following format:
    //origin target
    void print_hedge(std::string file_name){
        std::cout<<"Print halfedges"<<std::endl;
        std::ofstream file;
        file.open(file_name);
        int n_frontier_edges = 0;
        for(std::size_t i = 0; i < frontier_edges.size(); i++){
            if(frontier_edges[i] == true){
                n_frontier_edges++;
            }
        }
        file<<n_frontier_edges<<std::endl;
        for(std::size_t i = 0; i < mesh_input->halfEdges(); i++){
            if(frontier_edges[i] == true){
                file<<mesh_input->origin(i)<<" "<<mesh_input->target(i)<<"\n";
            }
        }
        file.close();
    }

    //Return a polygon generated from a seed edge
    _polygon generate_polygon(int e)
    {   
        _polygon poly;
        //search next frontier-edge
        int e_init = search_frontier_edge(e);
        int v_init = mesh_input->origin(e_init);
        int e_curr = mesh_input->next(e_init);
        int v_curr = mesh_input->origin(e_curr);
        poly.push_back(v_curr);
        while(e_curr != e_init && v_curr != v_init)
        {   
            e_curr = search_frontier_edge(e_curr);  
            //select triangle that contains v_curr as origin
            e_curr = mesh_input->next(e_curr);
            v_curr = mesh_input->origin(e_curr);
            poly.push_back(v_curr);
        }
        return poly;
    }


private:

    //Return true is the edge is terminal-edge or terminal border edge, 
    //but it only selects one halfedge as terminal-edge, the halfedge with lowest index is selected
    bool is_seed_edge(int e){
        int twin = mesh_input->twin(e);

        bool is_terminal_edge = (mesh_input->is_interior_face(twin) &&  (max_edges[e] && max_edges[twin]) );
        bool is_terminal_border_edge = (mesh_input->is_border_face(twin) && max_edges[e]);

        return (is_terminal_edge && e < twin ) || is_terminal_border_edge;
    }




    //Label max edges of all triangles in the triangulation
    //input: edge e indicent to a triangle t
    //output: position of edge e in max_edges[e] is labeled as true
    int label_max_edge(const int e)
    {
        //Calculates the size of each edge of a triangle 
        double dist0 = mesh_input->distance(e);
        double dist1 = mesh_input->distance(mesh_input->next(e));
        double dist2 = mesh_input->distance(mesh_input->prev(e));
        //Find the longest edge of the triangle
        if(std::max({dist0, dist1, dist2}) == dist0)
            return e;
        else if(std::max({dist0, dist1, dist2}) == dist1)
            return mesh_input->next(e);
        else
            return mesh_input->prev(e);
        return -1;
    }

 
    //Return true if the edge e is the lowest edge both triangles incident to e
    //in case of border edges, they are always labeled as frontier-edge
    bool is_frontier_edge(const int e)
    {
        int twin = mesh_input->twin(e);
        bool is_border_edge = mesh_input->is_border_face(e) || mesh_input->is_border_face(twin);
        bool is_not_max_edge = !(max_edges[e] || max_edges[twin]);
        return is_border_edge || is_not_max_edge;
    }

    //Travel in CCW order around the edges of vertex v from the edge e looking for the next frontier edge
    int search_frontier_edge(const int e)
    {
        int nxt = e;
        while(!frontier_edges[nxt])
            nxt = mesh_input->CW_edge_to_vertex(nxt);
        return nxt;
    }

    //return true if the polygon is not simple
    bool has_BarrierEdgeTip(int e_init){

        int e_curr = mesh_output->next(e_init);
        //travel inside frontier-edges of polygon
        while(e_curr != e_init){   
            //if the twin of the next halfedge is the current halfedge, then the polygon is not simple
            if( mesh_output->twin(mesh_output->next(e_curr)) == e_curr)
                return true;
            //travel to next half-edge
            e_curr = mesh_output->next(e_curr);
        }
        return false;
    }   

    //generate a polygon from a seed edge
    //input: Seed-edge
    //Output: seed frontier-edge of new popygon
    int travel_triangles(const int e)
    {   
        //search next frontier-edge
        int e_init = search_frontier_edge(e);
        //first frontier-edge is store to calculate the prev of next frontier-edfge
        int e_prev = e_init; 
        int v_init = mesh_input->origin(e_init);

        int e_curr = mesh_input->next(e_init);
        int v_curr = mesh_input->origin(e_curr);
        
        //travel inside frontier-edges of polygon
        while(e_curr != e_init && v_curr != v_init){   
            e_curr = search_frontier_edge(e_curr);

            //update next of previous frontier-edge
            mesh_output->set_next(e_prev, e_curr);  
            //update prev of current frontier-edge
            mesh_output->set_prev(e_curr, e_prev);

            //travel to next half-edge
            e_prev = e_curr;
            e_curr = mesh_input->next(e_curr);
            v_curr = mesh_input->origin(e_curr);
        }
        mesh_output->set_next(e_prev, e_init);
        mesh_output->set_prev(e_init, e_prev);
        return e_init;
    }
    
    //Given a barrier-edge tip v, return the middle edge incident to v
    //The function first calculate the degree of v - 1 and then divide it by 2, after travel to until the middle-edge
    //input: vertex v
    //output: edge incident to v
    int calculate_middle_edge(const int v){
        int frontieredge_with_bet = this->search_frontier_edge(mesh_input->edge_of_vertex(v));
        int internal_edges =mesh_input->degree(v) - 1; //internal-edges incident to v
        int adv = (internal_edges%2 == 0) ? internal_edges/2 - 1 : internal_edges/2 ;
        int nxt = mesh_input->CW_edge_to_vertex(frontieredge_with_bet);
        //back to traversing the edges of v_bet until select the middle-edge
        while (adv != 0){
            nxt = mesh_input->CW_edge_to_vertex(nxt);
            adv--;
        }
        return nxt;
    }

    //Given a seed edge e that generated polygon, split the polygon until remove al barrier-edge tips
    //input: seed edge e, polygon poly
    //output: polygon without barrier-edge tips
    void barrieredge_tip_reparation(const int e)
    {
        this->n_polygons_to_repair++;
        int x, y, i;
        int t1, t2;
        int middle_edge, v_bet;

        int e_init = e;
        int e_curr = mesh_output->next(e_init);
        //search by barrier-edge tips
        while(e_curr != e_init){   
            //if the twin of the next halfedge is the current halfedge, then the polygon is not simple
            if( mesh_output->twin(mesh_output->next(e_curr)) == e_curr){
                //std::cout<<"e_curr "<<e_curr<<" e_next "<<mesh_output->next(e_curr)<<" next del next "<<mesh_output->next(mesh_output->next(e_curr))<<" twin curr "<<mesh_output->twin(e_curr)<<" twin next "<<mesh_output->twin(mesh_output->next(e_curr))<<std::endl;

                n_barrier_edge_tips++;
                n_frontier_edges+=2;

                //select edge with bet
                v_bet = mesh_output->target(e_curr);
                middle_edge = calculate_middle_edge(v_bet);

                //middle edge that contains v_bet
                t1 = middle_edge;
                t2 = mesh_output->twin(middle_edge);
                
                //edges of middle-edge are labeled as frontier-edge
                this->frontier_edges[t1] = true;
                this->frontier_edges[t2] = true;

                //edges are use as seed edges and saves in a list
                triangle_list.push_back(t1);
                triangle_list.push_back(t2);

                seed_bet_mark[t1] = true;
                seed_bet_mark[t2] = true;
            }
                
            //travel to next half-edge
            e_curr = mesh_output->next(e_curr);
        }

        int t_curr;
        //generate polygons from seeds,
        //two seeds can generate the same polygon
        //so the bit_vector seed_bet_mark is used to label as false the edges that are already used
        int new_polygon_seed;
        while (!triangle_list.empty()){
            t_curr = triangle_list.back();
            triangle_list.pop_back();
            if(seed_bet_mark[t_curr]){
                this->n_polygons_added_after_repair++;
                seed_bet_mark[t_curr] = false;
                new_polygon_seed = generate_repaired_polygon(t_curr, seed_bet_mark);
                //Store the polygon in the as part of the mesh
                output_seeds.push_back(new_polygon_seed);
            }
        }

    }


    //Generate a polygon from a seed-edge and remove repeated seed from seed_list
    //POSIBLE BUG: el algoritmo no viaja por todos los halfedges dentro de un poligono, 
    //por lo que pueden haber semillas que no se borren y tener poligonos repetidos de output
    int generate_repaired_polygon(const int e, bit_vector &seed_list)
    {   
        int e_init = e;
        //search next frontier-edge
        while(!frontier_edges[e_init]){
            e_init = mesh_input->CW_edge_to_vertex(e_init);
            seed_list[e_init] = false; 
            //seed_list[mesh_input->twin(e_init)] = false;
        }        
        //first frontier-edge is store to calculate the prev of next frontier-edfge
        int e_prev = e_init; 
        int v_init = mesh_input->origin(e_init);

        int e_curr = mesh_input->next(e_init);
        int v_curr = mesh_input->origin(e_curr);
        seed_list[e_curr] = false;

        //travel inside frontier-edges of polygon
        while(e_curr != e_init && v_curr != v_init){   
            while(!frontier_edges[e_curr])
            {
                e_curr = mesh_input->CW_edge_to_vertex(e_curr);
                seed_list[e_curr] = false;
          //      seed_list[mesh_input->twin(e_curr)] = false;
            } 

            //update next of previous frontier-edge
            mesh_output->set_next(e_prev, e_curr);  
            //update prev of current frontier-edge
            mesh_output->set_prev(e_curr, e_prev);

            //travel to next half-edge
            e_prev = e_curr;        
            e_curr = mesh_input->next(e_curr);
            v_curr = mesh_input->origin(e_curr);
            seed_list[e_curr] = false;
            //seed_list[mesh_input->twin(e_curr)] = false;
        }
        mesh_output->set_next(e_prev, e_init);
        mesh_output->set_prev(e_init, e_prev);
        return e_init;
    }
};

#endif